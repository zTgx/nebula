#include <iostream>
#include <hip/hip_runtime.h> 

// The __global__ qualifier indicates that this function runs on the device (GPU) and can be called from the host (CPU).
// __global__ declaration specifier, marking it as a function that runs on the GPU but can be called from the host and executed in parallel.
__global__ void kernel() {

    // threadIdx.x is a built-in variable in CUDA that provides the thread ID within a block. 
    // blockIdx.x is a built-in variable that provides the block ID within the grid.
    printf("Hello from GPU thread %d in block %d\n", 
           threadIdx.x, blockIdx.x);
}

#define N 512 
 
__global__ void add(int *a, int *b, int *c) { 
   int index = threadIdx.x; 
   c[index] = a[index] + b[index]; 
} 

#include <hipblas.h> 
 
void cublasExample() { 
   hipblasHandle_t handle; 
   hipblasCreate(&handle); 
 
   float alpha = 1.0f; 
   float beta = 0.0f; 
   int NN = 1024;
 
   float* d_A; 
   float* d_B; 
   float* d_C; 
   hipMalloc((void**)&d_A, NN * NN * sizeof(float)); 
   hipMalloc((void**)&d_B, NN * NN * sizeof(float)); 
   hipMalloc((void**)&d_C, NN * NN * sizeof(float)); 
 
   hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_A, N, d_B, N, &beta, d_C, NN); 
 
   hipblasDestroy(handle); 
   hipFree(d_A); 
   hipFree(d_B); 
   hipFree(d_C); 
}

__global__ void addx(int *a, int *b, int *c) {
   *c = *a + *b;

   // This function acts as a barrier at which all threads in a block must wait until every thread reaches that point in the code. 
   __syncthreads();

   // __syncthreads(); // Ensure all additions are complete before writing back 

}

int main() {
    // Get device info
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    
    std::cout << "Launching on: " << prop.name << "\n";
    
    // Launch 4 parallel thread blocks
    // The first 2 indicates that the kernel will be launched with 2 blocks.
    // The second 2 indicates that each block will contain 2 threads.
    // <numBlocks, threadsPerBlock>
    kernel<<<2, 2>>>();

    // ensures the CPU waits for the GPU to complete execution before terminating the program.
    hipDeviceSynchronize();

    std::cout << "Done!\n";

    int a[N], b[N], c[N]; 
    int *d_a, *d_b, *d_c; 
  
    // allocates memory on the GPU.
    hipMalloc((void **) &d_a, N * sizeof(int)); 
    hipMalloc((void **) &d_b, N * sizeof(int)); 
    hipMalloc((void **) &d_c, N * sizeof(int)); 
  
    for (int i = 0; i < N; i++) { 
       a[i] = i; 
       b[i] = i * i; 
    } 
  
    // copies data from the host (CPU) to the device (GPU).
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice); 
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice); 
  
    // Launches the kernel with 1 block and N threads.
    add<<<1, N>>>(d_a, d_b, d_c); 
  
    // copies the result from the device (GPU) back to the host (CPU).
    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost); 
  
    for (int i = 0; i < N; i++) { 
       printf("%d + %d = %d\n", a[i], b[i], c[i]); 
    } 
  
    hipFree(d_a); hipFree(d_b); hipFree(d_c); 

    std::cout << "CUDA example completed.\n";

    cublasExample();
    std::cout << "cuBLAS example completed.\n";
    
   hipError_t err = hipGetLastError(); 
   if (err != hipSuccess) { 
      printf("CUDA Kernel launch failed: %s\n", hipGetErrorString(err)); 
   }

   return 0;
}